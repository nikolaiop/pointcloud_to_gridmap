#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

void initialize_matrix_cpu(float* matrix, int rows, int cols);
__global__ void initialize_matrix_gpu(float* matrix, int rows, int cols);

int main() {
    const int rows = 100000;
    const int cols = 100000;
    float* matrix_cpu = new float[rows * cols];
    float* matrix_gpu;

    // Initialize the matrix on the CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    initialize_matrix_cpu(matrix_cpu, rows, cols);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_cpu = end_cpu - start_cpu;
    std::cout << "CPU execution time: " << elapsed_cpu.count() << " ms\n";

    // Allocate memory on the GPU
    hipMalloc((void**)&matrix_gpu, rows * cols * sizeof(float));

    // Initialize the matrix on the GPU
    dim3 block_dim(32, 32);
    dim3 grid_dim((cols + block_dim.x - 1) / block_dim.x, (rows + block_dim.y - 1) / block_dim.y);
    auto start_gpu = std::chrono::high_resolution_clock::now();
    initialize_matrix_gpu<<<grid_dim, block_dim>>>(matrix_gpu, rows, cols);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_gpu = end_gpu - start_gpu;
    std::cout << "GPU execution time: " << elapsed_gpu.count() << " ms\n";

    // Free the memory on the GPU
    hipFree(matrix_gpu);

    // Free the memory on the CPU
    delete[] matrix_cpu;

    return 0;
}

void initialize_matrix_cpu(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = 1.0f; // Initialize the value to 1
    }
}

__global__ void initialize_matrix_gpu(float* matrix, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        matrix[index] = 1.0f; // Initialize the value to 1
    }
}